#include "hip/hip_runtime.h"
// C standard header files

#include <hiprand/hiprand_kernel.h>
#include <stdbool.h>
#include <stdio.h>
#include <stdlib.h>

const int LENGHT = 10;
const size_t SIZE_OF_RANDS = LENGHT * sizeof(double);

typedef struct Handle {
	bool gpu;bool help;bool verbose;
} Handle;

void fillRandomCPU(double* array, const int length);
__global__ void fillRandomGPU(unsigned int seed, double* array,
		const int length);
void printArray(const double* array, const int length);
void processParameters(Handle* handle, int argc, char* argv[]);

int main(int argc, char* argv[]) {
	Handle handle = {.gpu = false, .help = false, .verbose = false};
	processParameters(&handle, argc, argv);

	double* randomNumbers = (double*) malloc(SIZE_OF_RANDS);

	if (handle.gpu) {
		double* d_randomNumbers;
		hipMalloc((void **) &d_randomNumbers, SIZE_OF_RANDS);

		fillRandomGPU<<<1, 1>>>(time(NULL), d_randomNumbers, LENGHT);

		hipMemcpy(randomNumbers, d_randomNumbers, SIZE_OF_RANDS,
				hipMemcpyDeviceToHost);

		hipFree(d_randomNumbers);
	} else {
		fillRandomCPU(randomNumbers, LENGHT);
	}

	printArray(randomNumbers, LENGHT);

	free(randomNumbers);
}

void fillRandomCPU(double* array, const int length) {
	for (int i = 0; i < length; i++) {
		array[i] = (double) rand() / RAND_MAX;
	}
}

__global__ void fillRandomGPU(unsigned int seed, double* result,
		const int lenght) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	hiprandState_t state;
	hiprand_init(seed, id, id, &state);
	for (int i = 0; i < lenght; i++) {
		result[i] = hiprand(&state);
	}
}

void printArray(const double* array, const int length) {
	for (int i = 0; i < length; i++) {
		printf("array[%d] = %f\n", i, array[i]);
	}
}

/*
 * process the command line parameters and return a Handle struct with them
 */
void processParameters(Handle* handle, int argc, char* argv[]) {
	for (int currentArgument = 1; currentArgument < argc; currentArgument++) {
		switch (argv[currentArgument][1]) {
		case 'g':
			// switch to GPU version
			handle->gpu = true;
			break;
		case 'h':
			// print help message
			printf(
					"Parameters:\n"
							"\t-g\t\trun on the GPU\n"
							"\t-h\t\tprint this help message\n"
							"\t-v\t\tprint more information\n"
							"\nThis program is distributed under the terms of the LGPLv3 license\n");
			handle->help = true;
			exit(EXIT_SUCCESS);
			break;
		case 'v':
			// print more information
			handle->verbose = true;
			break;
		default:
			fprintf(stderr, "Wrong parameter: %s\n"
					"-h for help\n", argv[currentArgument]);
			exit(EXIT_FAILURE);
		}
	}
}
